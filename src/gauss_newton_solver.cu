#include "hip/hip_runtime.h"
#pragma once

#include "gauss_newton_solver.h"
#include "util.h"
#include "jacobian_util.h"
#include "device_util.h"
#include "device_array.h"
#include "opencv2/imgproc/imgproc.hpp"
#include "opencv2/highgui/highgui.hpp"

//#define TEST_TEXTURE

__global__ void cuComputeJacobian(
	//shared memory
	FaceBoundingBox face_bb,
	const int nFeatures, const int imageWidth, const int imageHeight,
	const int nFaceCoeffs, const int nPixels, const int n,
	const int nShapeCoeffs, const int nExpressionCoeffs, const int nAlbedoCoeffs,
	const int nUnknowns, const int nResiduals,
	const int nVerticesTimes3, const int nShapeCoeffsTotal, const int nExpressionCoeffsTotal, const int nAlbedoCoeffsTotal,
	const float wSparse, const float wDense, const float wReg,

	uchar* image, float* debug_frame,

	glm::mat4 face_pose, glm::mat3 drx, glm::mat3 dry, glm::mat3 drz, glm::mat4 projection, Eigen::Matrix3f jacobian_local,

	//device memory input
	int* prior_local_ids, glm::vec3* current_face, glm::vec2* sparse_features,

	float* p_shape_basis,
	float* p_expression_basis,
	float* p_albedo_basis,

	float* p_coefficients_shape,
	float* p_coefficients_expression,
	float* p_coefficients_albedo,
	float* p_coefficients_sh,

	hipTextureObject_t rgb,
	hipTextureObject_t barycentrics,
	hipTextureObject_t vertex_ids,

	//device memory output
	float* p_jacobian, float* p_residuals)
{
	int i = util::getThreadIndex1D();
	if (i >= n)
	{
		return;
	}

	Eigen::Map<Eigen::MatrixXf> jacobian(p_jacobian, nResiduals, nUnknowns);
	Eigen::Map<Eigen::VectorXf> residuals(p_residuals, nResiduals);

	Eigen::Map<Eigen::MatrixXf> shape_basis(p_shape_basis, nVerticesTimes3, nShapeCoeffsTotal);
	Eigen::Map<Eigen::MatrixXf> expression_basis(p_expression_basis, nVerticesTimes3, nExpressionCoeffsTotal);
	Eigen::Map<Eigen::MatrixXf> albedo_basis(p_albedo_basis, nVerticesTimes3, nAlbedoCoeffsTotal);

	// Regularization terms
	if (i >= nFeatures + nPixels)
	{
		int offset_rows = nFeatures * 2 + nPixels * 3;
		int offset_cols = 7;

		const int current_index = i - nFeatures - nPixels;
		const int expression_shift = nShapeCoeffs;
		const int albedo_shift = nShapeCoeffs + nExpressionCoeffs;

		float coefficient = 0.0f;
		int relative_index = current_index;

		// Shape
		if (current_index < expression_shift)
		{
			coefficient = p_coefficients_shape[relative_index];
		}
		// Expression
		else if (current_index < albedo_shift)
		{
			offset_rows += expression_shift;
			offset_cols += expression_shift;
			relative_index -= expression_shift;

			coefficient = p_coefficients_expression[relative_index];
		}
		// Albedo
		else
		{
			offset_rows += albedo_shift;
			offset_cols += albedo_shift;
			relative_index -= albedo_shift;

			coefficient = p_coefficients_albedo[relative_index];
		}

		jacobian(offset_rows + relative_index, offset_cols + relative_index) = wReg;
		residuals(offset_rows + relative_index) = coefficient * wReg;

		return;
	}

	// Dense terms
	if (i >= nFeatures)
	{
		int offset_rows = nFeatures * 2;

		const int current_index = i - nFeatures;
		unsigned int xp = current_index % face_bb.width + face_bb.x_min;
		unsigned int yp = current_index / face_bb.width + face_bb.y_min;

		int background_index = 3 * (xp + yp * imageWidth);
		int ygl = imageHeight - 1 - yp; // "height - 1 - index.y" OpenGL uses left-bottom corner as texture origin.
		float4 rgb_sampled = tex2D<float4>(rgb, xp, ygl);

		if (rgb_sampled.w < 1.0f) // pixel is not covered by face
		{
			return;
		}

		float4 barycentrics_sampled = tex2D<float4>(barycentrics, xp, ygl);
		int4 vertex_ids_sampled = tex2D<int4>(vertex_ids, xp, ygl);
		Eigen::Map<Eigen::Vector3f> face_rgb(reinterpret_cast<float*>(&rgb_sampled));
		Eigen::Vector3f frame_rgb;

		frame_rgb.x() = image[background_index] / 255.0f;
		frame_rgb.y() = image[background_index + 1] / 255.0f;
		frame_rgb.z() = image[background_index + 2] / 255.0f;

		Eigen::Vector3f residual = face_rgb - frame_rgb;
		residuals.block(offset_rows + current_index * 3, 0, 3, 1) = residual * wDense;

		//Albedo
		jacobian.block(offset_rows + current_index * 3, 7 + nShapeCoeffs + nExpressionCoeffs, 3, nAlbedoCoeffs) =
			(barycentrics_sampled.w * wDense * barycentrics_sampled.x) * albedo_basis.block(3 * vertex_ids_sampled.x, 0, 3, nAlbedoCoeffs) +
			(barycentrics_sampled.w * wDense * barycentrics_sampled.y) * albedo_basis.block(3 * vertex_ids_sampled.y, 0, 3, nAlbedoCoeffs) +
			(barycentrics_sampled.w * wDense * barycentrics_sampled.z) * albedo_basis.block(3 * vertex_ids_sampled.z, 0, 3, nAlbedoCoeffs);

		//SH
		auto number_of_vertices = nVerticesTimes3 / 3;
		auto albedos = current_face + number_of_vertices;
		auto normals = current_face + 2 * number_of_vertices;

		auto normal_a_unnorm_glm = glm::mat3(face_pose) * normals[vertex_ids_sampled.x];
		auto normal_b_unnorm_glm = glm::mat3(face_pose) * normals[vertex_ids_sampled.y];
		auto normal_c_unnorm_glm = glm::mat3(face_pose) * normals[vertex_ids_sampled.z];

		auto normal_a_glm = glm::normalize(normal_a_unnorm_glm);
		auto normal_b_glm = glm::normalize(normal_b_unnorm_glm);
		auto normal_c_glm = glm::normalize(normal_c_unnorm_glm);

		auto albedo_glm = barycentrics_sampled.x * albedos[vertex_ids_sampled.x] + barycentrics_sampled.y * albedos[vertex_ids_sampled.y] + barycentrics_sampled.z * albedos[vertex_ids_sampled.z];
		auto normal_unnorm_glm = barycentrics_sampled.x * normal_a_glm + barycentrics_sampled.y * normal_b_glm + barycentrics_sampled.z * normal_c_glm;
		auto normal_glm = glm::normalize(normal_unnorm_glm);

		Eigen::Vector3f albedo;
		albedo << albedo_glm.x, albedo_glm.y, albedo_glm.z;

		Eigen::Matrix<float, 1, 9> bands(9);
		bands(0, 0) = 1.0f;
		bands(0, 1) = normal_glm.y;
		bands(0, 2) = normal_glm.z;
		bands(0, 3) = normal_glm.x;
		bands(0, 4) = normal_glm.x * normal_glm.y;
		bands(0, 5) = normal_glm.y * normal_glm.z;
		bands(0, 6) = 3.0f * normal_glm.z * normal_glm.z - 1.0f;
		bands(0, 7) = normal_glm.x * normal_glm.z;
		bands(0, 8) = normal_glm.x * normal_glm.x - normal_glm.y * normal_glm.y;

		jacobian.block(offset_rows + current_index * 3, 7 + nShapeCoeffs + nExpressionCoeffs + nAlbedoCoeffs, 3, 9) = wDense * albedo * bands;

		//Pose
		Eigen::Matrix<float, 1, 3> dlight_dnormal;
		jacobian_util::computeDLightDNormal(dlight_dnormal, normal_glm, p_coefficients_sh);

		Eigen::Matrix<float, 3, 3> dnormal_dunnormnormal;
		jacobian_util::computeNormalizationJacobian(dnormal_dunnormnormal, normal_unnorm_glm);

		Eigen::Matrix<float, 3, 3> unnormnormal_jacobian = albedo * dlight_dnormal * dnormal_dunnormnormal;

		/*Eigen::Matrix<float, 3, 3> dnormal_dunnormnormal_sum = Eigen::MatrixXf::Zero(3, 3);
		//For 1st vertex normal
		jacobian_util::computeNormalizationJacobian(dnormal_dunnormnormal, normal_a_unnorm_glm);
		dnormal_dunnormnormal_sum += barycentrics_sampled.x * dnormal_dunnormnormal;

		//For 2nd vertex normal
		jacobian_util::computeNormalizationJacobian(dnormal_dunnormnormal, normal_b_unnorm_glm);
		dnormal_dunnormnormal_sum += barycentrics_sampled.y * dnormal_dunnormnormal;

		//For 3rd vertex normal
		jacobian_util::computeNormalizationJacobian(dnormal_dunnormnormal, normal_c_unnorm_glm);
		dnormal_dunnormnormal_sum += barycentrics_sampled.z * dnormal_dunnormnormal;

		Eigen::Matrix<float, 3, 3> jacobian_rotation;
		auto dx = drx * normals[vertex_ids_sampled.x];
		auto dy = dry * normals[vertex_ids_sampled.y];
		auto dz = drz * normals[vertex_ids_sampled.z];
		jacobian_rotation <<
			dx[0], dy[0], dz[0],
			dx[1], dy[1], dz[1],
			dx[2], dy[2], dz[2];

		jacobian.block(offset_rows + current_index * 3, 1, 3, 3) = unnormnormal_jacobian * dnormal_dunnormnormal_sum * jacobian_rotation * wDense;*/

		//Shape and expression
		Eigen::Matrix<float, 3, 3> v0_jacobian;
		Eigen::Matrix<float, 3, 3> v1_jacobian;
		Eigen::Matrix<float, 3, 3> v2_jacobian;

		jacobian_util::computeNormalJacobian(v0_jacobian, v1_jacobian, v2_jacobian,
			current_face[vertex_ids_sampled.x], current_face[vertex_ids_sampled.y], current_face[vertex_ids_sampled.z]);

		unnormnormal_jacobian = wDense * unnormnormal_jacobian * jacobian_local;
		v0_jacobian = unnormnormal_jacobian * v0_jacobian;
		v1_jacobian = unnormnormal_jacobian * v1_jacobian;
		v2_jacobian = unnormnormal_jacobian * v2_jacobian;

		jacobian.block(offset_rows + current_index * 3, 7, 3, nShapeCoeffs) =
			v0_jacobian * shape_basis.block(3 * vertex_ids_sampled.x, 0, 3, nShapeCoeffs) +
			v1_jacobian * shape_basis.block(3 * vertex_ids_sampled.y, 0, 3, nShapeCoeffs) +
			v2_jacobian * shape_basis.block(3 * vertex_ids_sampled.z, 0, 3, nShapeCoeffs);

		jacobian.block(offset_rows + current_index * 3, 7 + nShapeCoeffs, 3, nExpressionCoeffs) =
			v0_jacobian * expression_basis.block(3 * vertex_ids_sampled.x, 0, 3, nExpressionCoeffs) +
			v1_jacobian * expression_basis.block(3 * vertex_ids_sampled.y, 0, 3, nExpressionCoeffs) +
			v2_jacobian * expression_basis.block(3 * vertex_ids_sampled.z, 0, 3, nExpressionCoeffs);

		//Below is the derivative with respect to source image (frame_rgb)
		//Pose
		auto local_coord = barycentrics_sampled.x * current_face[vertex_ids_sampled.x] +
			barycentrics_sampled.y * current_face[vertex_ids_sampled.y] +
			barycentrics_sampled.z * current_face[vertex_ids_sampled.z];

		auto world_coord = face_pose * glm::vec4(local_coord, 1.0f);
		auto proj_coord = projection * world_coord;

		//Derivative of source image with respect to (u,v)
		//TODO: Check for boundary for xp and yp
		Eigen::Matrix<float, 3, 2> jacobian_uv = Eigen::MatrixXf::Zero(3, 2);

		int background_index_left = 3 * (xp - 1 + yp * imageWidth);
		int background_index_right = 3 * (xp + 1 + yp * imageWidth);
		int background_index_up = 3 * (xp + (yp - 1) * imageWidth);
		int background_index_down = 3 * (xp + (yp + 1) * imageWidth);
		jacobian_uv(0, 0) = (image[background_index_right] / 255.0f - image[background_index_left] / 255.0f) * 0.25f * imageWidth;
		jacobian_uv(1, 0) = (image[background_index_right + 1] / 255.0f - image[background_index_left + 1] / 255.0f) *  0.25f * imageWidth;
		jacobian_uv(2, 0) = (image[background_index_right + 2] / 255.0f - image[background_index_left + 2] / 255.0f) *  0.25f * imageWidth;
		jacobian_uv(0, 1) = -(image[background_index_down] / 255.0f - image[background_index_up] / 255.0f) *  0.25f  * imageHeight;
		jacobian_uv(1, 1) = -(image[background_index_down + 1] / 255.0f - image[background_index_up + 1] / 255.0f) *  0.25f  * imageHeight;
		jacobian_uv(2, 1) = -(image[background_index_down + 2] / 255.0f - image[background_index_up + 2] / 255.0f) *  0.25f  * imageHeight;
		jacobian_uv = -jacobian_uv;

		//Jacobian for homogenization (AKA division by w)
		Eigen::Matrix<float, 2, 3> jacobian_proj = Eigen::MatrixXf::Zero(2, 3);
		auto one_over_wp = 1.0f / proj_coord.w;
		jacobian_proj(0, 0) = one_over_wp;
		jacobian_proj(0, 2) = -proj_coord.x * one_over_wp * one_over_wp;

		jacobian_proj(1, 1) = one_over_wp;
		jacobian_proj(1, 2) = -proj_coord.y * one_over_wp * one_over_wp;

		//Jacobian for projection
		Eigen::Matrix<float, 3, 3> jacobian_world = Eigen::MatrixXf::Zero(3, 3);
		jacobian_world(0, 0) = projection[0][0];
		jacobian_world(1, 1) = projection[1][1];
		jacobian_world(2, 2) = -1.0f;

		//Jacobian for intrinsics
		Eigen::Matrix<float, 3, 1> jacobian_intrinsics = Eigen::MatrixXf::Zero(3, 1);
		jacobian_intrinsics(0, 0) = world_coord.x;
		//jacobian.block<3, 1>(offset_rows + current_index * 3, 0) = jacobian_uv * jacobian_proj * jacobian_intrinsics * wDense;

		//Derivative of world coordinates with respect to rotation coefficients
		auto dx = drx * local_coord;
		auto dy = dry * local_coord;
		auto dz = drz * local_coord;

		Eigen::Matrix<float, 3, 6> jacobian_pose = Eigen::MatrixXf::Zero(3, 6);
		jacobian_pose(0, 3) = 1.0f;
		jacobian_pose(1, 4) = 1.0f;
		jacobian_pose(2, 5) = 1.0f;
		jacobian_pose(0, 0) = dx[0];
		jacobian_pose(1, 0) = dx[1];
		jacobian_pose(2, 0) = dx[2];
		jacobian_pose(0, 1) = dy[0];
		jacobian_pose(1, 1) = dy[1];
		jacobian_pose(2, 1) = dy[2];
		jacobian_pose(0, 2) = dz[0];
		jacobian_pose(1, 2) = dz[1];
		jacobian_pose(2, 2) = dz[2];

		auto jacobian_proj_world = jacobian_uv * jacobian_proj * jacobian_world;
		jacobian.block<3, 6>(offset_rows + current_index * 3, 1) = jacobian_proj_world * jacobian_pose * wDense;

		//Derivative of world coordinates with respect to local coordinates.
		//This is basically the rotation matrix.
		auto jacobian_proj_world_local = jacobian_proj_world * jacobian_local * wDense;

		//Derivative of local coordinates with respect to shape and expression parameters
		jacobian.block(offset_rows + current_index * 3, 7, 3, nShapeCoeffs) +=
			(jacobian_proj_world_local * barycentrics_sampled.x) * shape_basis.block(3 * vertex_ids_sampled.x, 0, 3, nShapeCoeffs) +
			(jacobian_proj_world_local * barycentrics_sampled.y) * shape_basis.block(3 * vertex_ids_sampled.y, 0, 3, nShapeCoeffs) +
			(jacobian_proj_world_local * barycentrics_sampled.z) * shape_basis.block(3 * vertex_ids_sampled.z, 0, 3, nShapeCoeffs);

		jacobian.block(offset_rows + current_index * 3, 7 + nShapeCoeffs, 3, nExpressionCoeffs) +=
			(jacobian_proj_world_local * barycentrics_sampled.x) * expression_basis.block(3 * vertex_ids_sampled.x, 0, 3, nExpressionCoeffs) +
			(jacobian_proj_world_local * barycentrics_sampled.y) * expression_basis.block(3 * vertex_ids_sampled.y, 0, 3, nExpressionCoeffs) +
			(jacobian_proj_world_local * barycentrics_sampled.z) * expression_basis.block(3 * vertex_ids_sampled.z, 0, 3, nExpressionCoeffs);



#ifdef TEST_TEXTURE
		if (rgb_sampled.w > 0)
		{
			auto uv = glm::vec2(proj_coord.x, proj_coord.y) / proj_coord.w;
			uv.y = -uv.y;
			uv = uv * 0.5f + 0.5f;
			int pixel_x = uv.x * imageWidth;
			int pixel_y = uv.y * imageHeight;

			auto a_current_index = pixel_x + pixel_y * imageWidth;
			debug_frame[a_current_index * 3] = face_rgb.x();
			debug_frame[a_current_index * 3 + 1] = face_rgb.y();
			debug_frame[a_current_index * 3 + 2] = face_rgb.z();

			/*debug_frame[current_index * 3] = frame_rgb.x();
			debug_frame[current_index * 3 + 1] = frame_rgb.y();
			debug_frame[current_index * 3 + 2] = frame_rgb.z();*/

			/*debug_frame[current_index * 3] = barycentrics_sampled.x;
			debug_frame[current_index * 3 + 1] = barycentrics_sampled.y;
			debug_frame[current_index * 3 + 2] = barycentrics_sampled.z;*/

			/*debug_frame[current_index * 3] = (normal_glm.x + 1.0f) * 0.5f;
			debug_frame[current_index * 3 + 1] = (normal_glm.y + 1.0f) * 0.5f;
			debug_frame[current_index * 3 + 2] = (normal_glm.z + 1.0f) * 0.5f;*/
		}
		else
		{
			//debug_frame[current_index * 3] = image[background_index] / 255.0f;
			//debug_frame[current_index * 3 + 1] = image[background_index + 1] / 255.0f;
			//debug_frame[current_index * 3 + 2] = image[background_index + 2] / 255.0f;
		}
#endif // TEST_TEXTURE
		return;
	}

	// Sparse terms
	auto vertex_id = prior_local_ids[i];
	auto local_coord = current_face[vertex_id];

	auto world_coord = face_pose * glm::vec4(local_coord, 1.0f);
	auto proj_coord = projection * world_coord;
	auto uv = glm::vec2(proj_coord.x, proj_coord.y) / proj_coord.w;

	//Residual
	auto residual = uv - sparse_features[i];

	residuals(i * 2) = residual.x * wSparse;
	residuals(i * 2 + 1) = residual.y * wSparse;

	//Jacobian for homogenization (AKA division by w)
	Eigen::Matrix<float, 2, 3> jacobian_proj = Eigen::MatrixXf::Zero(2, 3);
	auto one_over_wp = 1.0f / proj_coord.w;
	jacobian_proj(0, 0) = one_over_wp;
	jacobian_proj(0, 2) = -proj_coord.x * one_over_wp * one_over_wp;

	jacobian_proj(1, 1) = one_over_wp;
	jacobian_proj(1, 2) = -proj_coord.y * one_over_wp * one_over_wp;

	//Jacobian for projection
	Eigen::Matrix<float, 3, 3> jacobian_world = Eigen::MatrixXf::Zero(3, 3);
	jacobian_world(0, 0) = projection[0][0];
	jacobian_world(1, 1) = projection[1][1];
	jacobian_world(2, 2) = -1.0f;

	//Jacobian for intrinsics
	Eigen::Matrix<float, 3, 1> jacobian_intrinsics = Eigen::MatrixXf::Zero(3, 1);
	jacobian_intrinsics(0, 0) = world_coord.x;
	jacobian.block<2, 1>(i * 2, 0) = jacobian_proj * jacobian_intrinsics * wSparse;

	//Derivative of world coordinates with respect to rotation coefficients
	auto dx = drx * local_coord;
	auto dy = dry * local_coord;
	auto dz = drz * local_coord;

	Eigen::Matrix<float, 3, 6> jacobian_pose = Eigen::MatrixXf::Zero(3, 6);
	jacobian_pose(0, 3) = 1.0f;
	jacobian_pose(1, 4) = 1.0f;
	jacobian_pose(2, 5) = 1.0f;
	jacobian_pose(0, 0) = dx[0];
	jacobian_pose(1, 0) = dx[1];
	jacobian_pose(2, 0) = dx[2];
	jacobian_pose(0, 1) = dy[0];
	jacobian_pose(1, 1) = dy[1];
	jacobian_pose(2, 1) = dy[2];
	jacobian_pose(0, 2) = dz[0];
	jacobian_pose(1, 2) = dz[1];
	jacobian_pose(2, 2) = dz[2];

	auto jacobian_proj_world = jacobian_proj * jacobian_world * wSparse;
	jacobian.block<2, 6>(i * 2, 1) = jacobian_proj_world * jacobian_pose;

	//Derivative of world coordinates with respect to local coordinates.
	//This is basically the rotation matrix.
	auto jacobian_proj_world_local = jacobian_proj_world * jacobian_local;

	//Derivative of local coordinates with respect to shape and expression parameters
	//This is basically the corresponding (to unique vertices we have chosen) rows of basis matrices.
	auto jacobian_shape = jacobian_proj_world_local * shape_basis.block(3 * vertex_id, 0, 3, nShapeCoeffs);
	jacobian.block(i * 2, 7, 2, nShapeCoeffs) = jacobian_shape;

	auto jacobian_expression = jacobian_proj_world_local * expression_basis.block(3 * vertex_id, 0, 3, nExpressionCoeffs);
	jacobian.block(i * 2, 7 + nShapeCoeffs, 2, nExpressionCoeffs) = jacobian_expression;
		}

__global__ void cuComputeVisiblePixelsAndBB(hipTextureObject_t texture, FaceBoundingBox* face_bb, int width, int height)
{
	auto index = util::getThreadIndex2D();
	if (index.x >= width || index.y >= height)
	{
		return;
	}
	int y = height - 1 - index.y; // "height - 1 - index.y" is used since OpenGL uses left-bottom corner as texture origin.
	float4 color = tex2D<float4>(texture, index.x, y);

	if (color.w > 0.0f)
	{
		atomicInc(&face_bb->num_visible_pixels, UINT32_MAX);
		atomicMin(&face_bb->x_min, index.x);
		atomicMin(&face_bb->y_min, index.y);
		atomicMax(&face_bb->x_max, index.x);
		atomicMax(&face_bb->y_max, index.y);
	}
}

FaceBoundingBox GaussNewtonSolver::computeFaceBoundingBox(const int imageWidth, const int imageHeight)
{
	FaceBoundingBox bb;
	util::copy(m_face_bb, &bb, 1);

	//TODO: Arrange this (16,16) according to TitanX when we use it.
	dim3 threads_meta(16, 16);
	dim3 blocks_meta(imageWidth / threads_meta.x + 1, imageHeight / threads_meta.y + 1);

	cuComputeVisiblePixelsAndBB << <blocks_meta, threads_meta >> > (m_texture_rgb, m_face_bb.getPtr(), imageWidth, imageHeight);

	util::copy(&bb, m_face_bb, 1);
	//std::cout << bb.num_visible_pixels << " " << bb.x_min << " " << bb.y_min << " " << bb.x_max << " " << bb.y_max << std::endl;

	if (bb.num_visible_pixels <= 0 || bb.x_min >= bb.x_max || bb.y_min >= bb.y_max)
	{
		std::cout << "Warning: invalid face bounding box!" << std::endl;
	}

	bb.width = bb.x_max - bb.x_min;
	bb.height = bb.y_max - bb.y_min;

	return bb;
}

void GaussNewtonSolver::computeJacobian(
	//shared memory
	const FaceBoundingBox face_bb,
	const int nFeatures, const int imageWidth, const int imageHeight,
	const int nShapeCoeffs, const int nExpressionCoeffs, const int nAlbedoCoeffs,
	const int nUnknowns, const int nResiduals,
	const int nVerticesTimes3, const int nShapeCoeffsTotal, const int nExpressionCoeffsTotal, const int nAlbedoCoeffsTotal, const int nShcoeffsTotal,
	float sparseWeight, float denseWeight, float regularizationWeight,

	uchar* image,

	const glm::mat4& face_pose, const glm::mat3& drx, const glm::mat3& dry, const glm::mat3& drz, const glm::mat4& projection, const Eigen::Matrix3f& jacobian_local,

	//device memory input
	int* prior_local_ids, glm::vec3* current_face, glm::vec2* sparse_features,

	float* p_shape_basis,
	float* p_expression_basis,
	float* p_albedo_basis,

	float* p_coefficients_shape,
	float* p_coefficients_expression,
	float* p_coefficients_albedo,
	float* p_coefficients_sh,

	//device memory output
	float* p_jacobian, float* p_residuals
) const
{
	const int nPixels = face_bb.width * face_bb.height;

	const int nFaceCoeffs = nShapeCoeffs + nExpressionCoeffs + nAlbedoCoeffs;
	const int n = nFeatures + nPixels + nFaceCoeffs;

	//TODO: Fine tune these configs according to TitanX in the end.
	const int threads = 128;
	const int block = (n + threads - 1) / threads;

	util::DeviceArray<float> temp_memory(imageWidth * imageHeight * 3);

	auto time = util::runKernelGetExecutionTime([&]() {cuComputeJacobian << <block, threads >> > (
		//shared memory
		face_bb,
		nFeatures, imageWidth, imageHeight,
		nFaceCoeffs, nPixels, n,
		nShapeCoeffs, nExpressionCoeffs, nAlbedoCoeffs,
		nUnknowns, nResiduals,
		nVerticesTimes3, nShapeCoeffsTotal, nExpressionCoeffsTotal, nAlbedoCoeffsTotal,
		glm::sqrt(sparseWeight / nFeatures), glm::sqrt(denseWeight / face_bb.num_visible_pixels), glm::sqrt(regularizationWeight),

		image, temp_memory.getPtr(),

		face_pose, drx, dry, drz, projection, jacobian_local,

		//device memory input
		prior_local_ids, current_face, sparse_features,

		p_shape_basis,
		p_expression_basis,
		p_albedo_basis,

		p_coefficients_shape,
		p_coefficients_expression,
		p_coefficients_albedo,
		p_coefficients_sh,

		m_texture_rgb,
		m_texture_barycentrics,
		m_texture_vertex_ids,

		//device memory output
		p_jacobian, p_residuals
		);
		});
	std::cout << "Jacobian kernel time: " << time << std::endl;


	hipDeviceSynchronize();

#ifdef TEST_TEXTURE
	static cv::VideoWriter video_writer("../../out_debug.avi", cv::VideoWriter::fourcc('M', 'J', 'P', 'G'), 24, cv::Size(imageWidth, imageHeight));
	std::vector<float> temp_memory_host(temp_memory.getSize());
	util::copy(temp_memory_host, temp_memory, temp_memory.getSize());
	cv::Mat image_debug(cv::Size(imageWidth, imageHeight), CV_8UC3);
	for (int y = 0; y < image_debug.rows; y++)
	{
		for (int x = 0; x < image_debug.cols; x++)
		{
			auto idx = (x + y * imageWidth) * 3;
			// OpenCV expects it to be an BGRA image.
			image_debug.at<cv::Vec3b>(cv::Point(x, y)) = cv::Vec3b(255.0f * cv::Vec3f(temp_memory_host[idx + 2], temp_memory_host[idx + 1], temp_memory_host[idx]));
		}
	}
	video_writer.write(image_debug);
	//cv::imwrite("../../dense_test.png", image_debug);
#endif // TEST_TEXTURE
}

__global__ void cuComputeJTJDiagonals(const int nUnknowns, const int nCurrentResiduals, const int nResiduals, float* jacobian, float* diagJTJ)
{
	int tid = threadIdx.x;
	int col = blockIdx.x;

	float sum = 0.0f;
	for (int row = tid; row < nCurrentResiduals; row += blockDim.x)
	{
		auto v = jacobian[col * nResiduals + row];
		sum += v * v;
	}

	atomicAdd(&diagJTJ[col], sum);
}

__global__ void cuElementwiseMultiplication(float* v1, float* v2, float* out)
{
	int i = util::getThreadIndex1D();
	out[i] = v1[i] * v2[i];
}

__global__ void cuOneOverElement(float* diagJTJ, float* invJTJ)
{
	int i = util::getThreadIndex1D();

	invJTJ[i] = 1.0f / (glm::max(diagJTJ[i], 1.0e-4f));
}

void GaussNewtonSolver::computeDiagJTJ(const int nUnknowns, const int nCurrentResiduals, const int nResiduals, float* jacobian, float* diagJTJ)
{
	cuComputeJTJDiagonals << <nUnknowns, 128 >> > (nUnknowns, nCurrentResiduals, nResiduals, jacobian, diagJTJ);
	hipDeviceSynchronize();
}

void GaussNewtonSolver::computeInverseJTJ(const int nUnknowns, float* diagJTJ, float* invJTJ)
{
	cuOneOverElement << <1, nUnknowns >> > (diagJTJ, invJTJ);
	hipDeviceSynchronize();
}

void GaussNewtonSolver::elementwiseMultiplication(const int nElements, float* v1, float* v2, float* out)
{
	cuElementwiseMultiplication << <1, nElements >> > (v1, v2, out);
	hipDeviceSynchronize();
}